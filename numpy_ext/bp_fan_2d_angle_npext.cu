/*
 * @Description: implement numpy extension library functions
 * @Author: Tianling Lyu
 * @Date: 2021-01-09 20:13:54
 * @LastEditors: Please set LastEditors
 * @LastEditTime: 2021-01-11 18:14:51
 */

#include "numpy_ext/bp_fan_2d_angle_npext.h"
#include "numpy_ext/common.h"
#ifdef USE_CUDA
#include "hip/hip_runtime.h"
#endif

namespace np_ext {

#define FanBpAngleContainer OpContainer<FanBpAngleNPExt, FanBp2DAngleAllocParam, FanBp2DAngleRunParam>

FanBpAngleContainer fan_fp_angle_container_;

FanBpAngleNPExt::FanBpAngleNPExt(const FanBp2DAngleAllocParam& param)
    : param_(param.param), allocated_(false), bp_prep_(param.param), 
    bp_(param.param), xpos_(nullptr), ypos_(nullptr), sincostbl_(nullptr), angles_(param.angles)
{
#ifdef USE_CUDA
    in_ = nullptr;
    out_ = nullptr;
#endif
}

FanBpAngleNPExt::~FanBpAngleNPExt()
{
    if (device_ < 0)
    {
        // allocated on CPU
        if (xpos_ != nullptr)
            delete[] xpos_;
        if (ypos_ != nullptr)
            delete[] ypos_;
        if (sincostbl_ != nullptr)
            delete[] sincostbl_;
    }
    else
    {
        // allocated on GPU
#ifdef USE_CUDA
        if (xpos_ != nullptr)
            hipFree(xpos_);
        if (ypos_ != nullptr)
            hipFree(ypos_);
        if (sincostbl_ != nullptr)
            hipFree(sincostbl_);
        if (in_ != nullptr)
            hipFree(in_);
        if (out_ != nullptr)
            hipFree(out_);
#endif
    }
}

bool FanBpAngleNPExt::allocate() {
    if (allocated_) return true;
    if (device_ < 0) {
        xpos_ = new double[param_.nx];
        ypos_ = new double[param_.ny];
        sincostbl_ = new double[param_.na*2];
        allocated_ = true;
        return bp_prep_.calculate_on_cpu(angles_, xpos_, ypos_, sincostbl_);
    } else {
#ifdef USE_CUDA
        hipError_t err;
        err = hipSetDevice(device_);
        if (err != hipSuccess) 
            throw std::runtime_error("Device not found!");
        err = hipStreamCreate(&stream_);
        if (err != hipSuccess) 
            throw std::runtime_error("Stream initialization failed!");
        err = hipMalloc(&xpos_, param_.nx*sizeof(double));
        if (err != hipSuccess)
            throw std::runtime_error("CUDA allocate xpos failed!");
        err = hipMalloc(&ypos_, param_.ny*sizeof(double));
        if (err != hipSuccess)
            throw std::runtime_error("CUDA allocate ypos failed!");
        err = hipMalloc(&sincostbl_, param_.na*2*sizeof(double));
        if (err != hipSuccess)
            throw std::runtime_error("CUDA allocate sincostbl failed!");
        err = hipMalloc(&in_, param_.ns*param_.na*sizeof(double));
        if (err != hipSuccess)
            throw std::runtime_error("CUDA allocate input array failed!");
        err = hipMalloc(&out_, param_.nx*param_.ny*sizeof(double));
        if (err != hipSuccess)
            throw std::runtime_error("CUDA allocate output array failed!");
        double* angles_gpu;
        err = hipMalloc(&angles_gpu, param_.na*sizeof(double));
        if (err != hipSuccess)
            throw std::runtime_error("CUDA allocate angles array failed!");
        err = hipMemcpy(angles_gpu, angles_, param_.na*sizeof(double), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            hipFree(angles_gpu);
            throw std::runtime_error("CUDA memcpy angles array failed!");
        }
        allocated_ = true;
        return bp_prep_.calculate_on_gpu(angles_gpu, xpos_, ypos_, sincostbl_, stream_);
#else
        return false;
#endif
    }
}

bool FanBpAngleNPExt::run(const FanBp2DAngleRunParam& param)
{
    if (device_ < 0) {
        // use CPU
        return bp_.calculate_on_cpu(param.in, param.out, xpos_, ypos_, sincostbl_);
    } else {
        // use GPU
#ifdef USE_CUDA
        hipMemcpy(in_, param.in, param_.na*param_.ns*sizeof(double), hipMemcpyHostToDevice);
        bool finish = bp_.calculate_on_gpu(in_, out_, xpos_, ypos_, sincostbl_, stream_);
        hipMemcpy(param.out, out_, param_.nx*param_.ny*sizeof(double), hipMemcpyDeviceToHost);
        return finish;
#else
        return false;
#endif
    }
}

} // namespace np_ext

#if defined(_WIN32)
#define DLL_EXPORT _declspec(dllexport)
#else
#define DLL_EXPORT
#endif

DLL_EXPORT extern "C"
int fan_bp_2d_angle_create(double* angles, unsigned int ns, unsigned int na, 
    double ds, double offset_s, unsigned int nx, unsigned int ny, double dx, 
    double dy, double offset_x, double offset_y, double dso, double dsd, 
    double fov)
{
    np_ext::FanBp2DAngleAllocParam param(ns, na, ds, offset_s, nx, ny, dx, dy, 
        offset_x, offset_y, dso, dsd, fov, angles);
    int handle = np_ext::fan_fp_angle_container_.create(param);
    return handle;
}

DLL_EXPORT extern "C"
bool fan_bp_2d_angle_run(int handle, double* in, double* out)
{
    np_ext::FanBp2DAngleRunParam param(in, out);
    return np_ext::fan_fp_angle_container_.run(handle, param);
}

DLL_EXPORT extern "C"
bool fan_bp_2d_angle_destroy(int handle)
{
    return np_ext::fan_fp_angle_container_.erase(handle);
}