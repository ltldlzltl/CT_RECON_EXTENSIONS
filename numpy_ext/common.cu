/*
 * @Description: 
 * @Author: Tianling Lyu
 * @Date: 2021-01-09 18:06:32
 * @LastEditors: Please set LastEditors
 * @LastEditTime: 2021-01-11 18:04:07
 */

#include "numpy_ext/common.h"
#ifdef USE_CUDA
#include "hip/hip_runtime.h"
#endif

namespace np_ext {

int device_ = -1;
#ifdef USE_CUDA
hipStream_t stream_ = 0;
#endif

} // namespace np_ext

#if defined(_WIN32)
#define DLL_EXPORT _declspec(dllexport)
#else
#define DLL_EXPORT
#endif

DLL_EXPORT extern "C"
bool set_device(const int device)
{
    np_ext::device_ = device;
    if (np_ext::device_ >= 0) {
        // use GPU
#ifdef USE_CUDA
        hipError_t err;
        err = hipSetDevice(np_ext::device_);
        if (err != hipSuccess) 
            throw std::runtime_error("Device not found!");
        err = hipStreamCreate(&np_ext::stream_);
        if (err != hipSuccess) 
            throw std::runtime_error("Stream initialization failed!");
        return true;
#else
        return false;
#endif
    }
    return true;
}

DLL_EXPORT extern "C"
bool clear()
{
    if (np_ext::device_ >= 0) {
#ifdef USE_CUDA
        if (np_ext::stream_ != 0) 
            hipStreamDestroy(np_ext::stream_);
#endif
    }
    return true;
}