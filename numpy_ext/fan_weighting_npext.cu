/*
 * @Description: implement fan weighting numpy extension library functions
 * @Author: Tianling Lyu
 * @Date: 2021-01-10 22:35:16
 * @LastEditors: Please set LastEditors
 * @LastEditTime: 2021-01-11 18:01:09
 */

#include "numpy_ext/fan_weighting_npext.h"
#include "numpy_ext/common.h"
#ifdef USE_CUDA
#include "hip/hip_runtime.h"
#endif

#include <cstdio>

namespace np_ext {

#define FanWContainer OpContainer<FanWeightingNPExt, ct_recon::FanWeightingParam, FanWeightingRunParam>
FanWContainer fan_w_container_;

FanWeightingNPExt::FanWeightingNPExt(const ct_recon::FanWeightingParam& param, int device)
    : param_(param), device_(device), allocated_(false), fw_(param)
{
#ifdef USE_CUDA
    inout_ = nullptr;
    stream_ = nullptr;
#endif
}

FanWeightingNPExt::~FanWeightingNPExt()
{
    if (device_ >= 0)  {
#ifdef USE_CUDA
        if (inout_ != nullptr)
            hipFree(inout_);
        if (stream_ != nullptr)
            hipStreamDestroy(stream_);
#endif
    }
}

bool FanWeightingNPExt::allocate() {
    if (allocated_) return true;
    if (device_ < 0) {
        allocated_ = true;
        return true;
    } else {
#ifdef USE_CUDA
        hipError_t err;
        err = hipSetDevice(device_);
        if (err != hipSuccess) 
            throw std::runtime_error("Device not found!");
        err = hipStreamCreate(&stream_);
        if (err != hipSuccess) 
            throw std::runtime_error("Stream initialization failed!");
        err = hipMalloc(&inout_, param_.ns*param_.nrow*sizeof(double));
        if (err != hipSuccess)
            throw std::runtime_error("CUDA allocate inout array failed!");
        allocated_ = true;
        return true;
#else
        return false;
#endif
    }
}

bool FanWeightingNPExt::run(const FanWeightingRunParam& param)
{
    if (device_ < 0) {
        return fw_.calculate_on_cpu(param.in, param.out);
    } else {
        // use GPU
#ifdef USE_CUDA
        hipMemcpy(inout_, param.in, param_.nrow*param_.ns*sizeof(double), hipMemcpyHostToDevice);
        bool finish = fw_.calculate_on_gpu(inout_, inout_, stream_);
        hipMemcpy(param.out, inout_, param_.nrow*param_.ns*sizeof(double), hipMemcpyDeviceToHost);
        return finish;
#else
        return false;
#endif
    }
}

} // namespace np_ext

#if defined(_WIN32)
#define DLL_EXPORT _declspec(dllexport)
#else
#define DLL_EXPORT
#endif

#include <string>

DLL_EXPORT extern "C"
int fan_weighting_create(unsigned int ns, unsigned int nrow, double ds, 
    double dso, double dsd, int type, int device)
{
    std::string s_type;
    switch (type) {
        case 1: s_type = "fan"; break;
        case 2: s_type = "flat"; break;
        default: {
            throw std::runtime_error("Unknown filter type!");
        }
    }
    ct_recon::FanWeightingParam param(ns, nrow, ds, dso, dsd, s_type);
    int handle = np_ext::fan_w_container_.create(param, device);
    return handle;
}

DLL_EXPORT extern "C"
bool fan_weighting_run(int handle, double* in, double* out)
{
    np_ext::FanWeightingRunParam param(in, out);
    return np_ext::fan_w_container_.run(handle, param);
}

DLL_EXPORT extern "C"
bool fan_weighting_destroy(int handle)
{
    return np_ext::fan_w_container_.erase(handle);
}