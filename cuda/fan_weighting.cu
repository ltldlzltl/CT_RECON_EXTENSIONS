#include "hip/hip_runtime.h"
/*
 * @Description: 
 * @Author: Tianling Lyu
 * @Date: 2021-01-09 08:47:49
 * @LastEditors: Tianling Lyu
 * @LastEditTime: 2021-03-11 10:28:59
 */

 #include "include/fan_weighting.h"

#include <cstdio>
#include "cuda/cuda_common.h"

namespace ct_recon {
    #ifdef USE_CUDA
    template <typename T>
    __global__ void FlatWeightingKernel(const T* in, T* out, 
        const FanWeightingParam param, const int n_elements)
    {
        double cents = static_cast<double>(param.ns-1) / 2;
        for (int thread_id : CudaGridRangeX<int>(n_elements)) {
            int is = thread_id % param.ns;
            double s = param.ds * (static_cast<double>(is) - cents);
            double w = param.dso * fabs(cos(atan2(s, param.dsd))) / param.dsd;
            out[thread_id] = w * in[thread_id];
        }
        return;
    }

    template <typename T>
    __global__ void FanWeightingKernel(const T* in, T* out, 
        const FanWeightingParam param, const int n_elements)
    {
        double cents = static_cast<double>(param.ns-1) / 2;
        for (int thread_id : CudaGridRangeX<int>(n_elements)) {
            int is = thread_id % param.ns;
            double s = param.ds * (static_cast<double>(is) - cents);
            double w = param.dso * fabs(cos(s / param.dsd)) / param.dsd;
            out[thread_id] = w * in[thread_id];
        }
        return;
    }

    template <typename T>
    __global__ void FlatWeightingGradKernel(const T* in, T* out, 
        const FanWeightingParam param, const int n_elements)
    {
        double cents = static_cast<double>(param.ns-1) / 2;
        for (int thread_id : CudaGridRangeX<int>(n_elements)) {
            int is = thread_id % param.ns;
            double s = param.ds * (static_cast<double>(is) - cents);
            double w = param.dsd * (param.dso * fabs(cos(atan2(s, param.dsd))));
            out[thread_id] = in[thread_id] * w;
        }
        return;
    }

    template <typename T>
    __global__ void FanWeightingGradKernel(const T* in, T* out, 
        const FanWeightingParam param, const int n_elements)
    {
        double cents = static_cast<double>(param.ns-1) / 2;
        for (int thread_id : CudaGridRangeX<int>(n_elements)) {
            int is = thread_id % param.ns;
            double s = param.ds * (static_cast<double>(is) - cents);
            double w = param.dsd * (param.dso * fabs(cos(s / param.dsd)));
            out[thread_id] = in[thread_id] * w;
        }
        return;
    }

    template <>
    bool FanWeighting<float>::calculate_on_gpu(const float* in, 
        float* out, hipStream_t stream) const
    {
        int n_elements = param_.ns*param_.nrow;
        CudaLaunchConfig config = GetCudaLaunchConfig(n_elements);
        if (param_.type == 1) {
            FanWeightingKernel<float>
                <<<config.block_count, config.thread_per_block, 0, stream>>>
                (in, out, param_, n_elements);
        } else if (param_.type == 2) {
            FlatWeightingKernel<float>
                <<<config.block_count, config.thread_per_block, 0, stream>>>
                (in, out, param_, n_elements);
        } else {
            return false;
        }
        hipError_t err = hipDeviceSynchronize();
        return err==hipSuccess;
    }

    template <>
    bool FanWeighting<double>::calculate_on_gpu(const double* in, 
        double* out, hipStream_t stream) const
    {
        int n_elements = param_.ns*param_.nrow;
        CudaLaunchConfig config = GetCudaLaunchConfig(n_elements);
        if (param_.type == 1) {
            FanWeightingKernel<double>
                <<<config.block_count, config.thread_per_block, 0, stream>>>
                (in, out, param_, n_elements);
        } else if (param_.type == 2) {
            FlatWeightingKernel<double>
                <<<config.block_count, config.thread_per_block, 0, stream>>>
                (in, out, param_, n_elements);
        } else {
            return false;
        }
        hipError_t err = hipDeviceSynchronize();
        return err==hipSuccess;
    }

    template <>
    bool FanWeightingGrad<float>::calculate_on_gpu(const float* in, 
        float* out, hipStream_t stream) const
    {
        int n_elements = param_.ns*param_.nrow;
        CudaLaunchConfig config = GetCudaLaunchConfig(n_elements);
        if (param_.type == 1) {
            FanWeightingGradKernel<float>
                <<<config.block_count, config.thread_per_block, 0, stream>>>
                (in, out, param_, n_elements);
        } else if (param_.type == 2) {
            FlatWeightingGradKernel<float>
                <<<config.block_count, config.thread_per_block, 0, stream>>>
                (in, out, param_, n_elements);
        } else {
            return false;
        }
        hipError_t err = hipDeviceSynchronize();
        return err==hipSuccess;
    }

    template <>
    bool FanWeightingGrad<double>::calculate_on_gpu(const double* in, 
        double* out, hipStream_t stream) const
    {
        int n_elements = param_.ns*param_.nrow;
        CudaLaunchConfig config = GetCudaLaunchConfig(n_elements);
        if (param_.type == 1) {
            FanWeightingGradKernel<double>
                <<<config.block_count, config.thread_per_block, 0, stream>>>
                (in, out, param_, n_elements);
        } else if (param_.type == 2) {
            FlatWeightingGradKernel<double>
                <<<config.block_count, config.thread_per_block, 0, stream>>>
                (in, out, param_, n_elements);
        } else {
            return false;
        }
        hipError_t err = hipDeviceSynchronize();
        return err==hipSuccess;
    }

    #endif // USE_CUDA
} // namespace ct_recon